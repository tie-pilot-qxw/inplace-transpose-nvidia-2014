#include "hip/hip_runtime.h"
#include "introspect.cuh"
#include "../common/index.cuh"
#include "../common/gcd.h"
#include "../common/reduced_math.cuh"
#include "equations.cuh"
#include "smem.cuh"
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>


namespace inplace {
namespace detail {

namespace c2r {

struct fused_preop {
    reduced_divisor m;
    reduced_divisor b;
    __host__  fused_preop(int _m, int _b) : m(_m), b(_b) {}
    __host__ __device__
    int operator()(const int& i, const int& j) {
        return (int)m.mod(i + (int)b.div(j));
    }
};

//This shuffler exists for cases where m, n are large enough to cause overflow
struct long_shuffle {
    int m, n, k;
    reduced_divisor_64 b;
    reduced_divisor c;
    __host__
    long_shuffle(int _m, int _n, int _c, int _k) : m(_m), n(_n), k(_k),
                                                   b(_n/_c), c(_c) {}
    int i;
    __host__ __device__ 
    void set_i(const int& _i) {
        i = _i;
    }
    __host__ __device__
    int f(const int& j) {
        int r = j + i * (n - 1);
        //The (int) casts here prevent unsigned promotion
        //and the subsequent underflow: c implicitly casts
        //int - unsigned int to
        //unsigned int - unsigned int
        //rather than to
        //int - int
        //Which leads to underflow if the result is negative.
        if (i - (int)c.mod(j) <= m - (int)c.get()) {
            return r;
        } else {
            return r + m;
        }
    }
    
    __host__ __device__
    int operator()(const int& j) {
        int fij = f(j);
        unsigned int fijdivc, fijmodc;
        c.divmod(fij, fijdivc, fijmodc);
        int term_1 = b.mod((long long)k * (long long)fijdivc);
        int term_2 = ((int)fijmodc) * (int)b.get();
        return term_1+term_2;
    }
};

struct fused_postop {
    reduced_divisor m;
    int n, c;
    __host__ 
    fused_postop(int _m, int _n, int _c) : m(_m), n(_n), c(_c) {}
    __host__ __device__
    int operator()(const int& i, const int& j) {
        return (int)m.mod(i * n - (int)m.div(i * c) + j);
    }
};


}

namespace r2c {

struct fused_preop {
    reduced_divisor a;
    reduced_divisor c;
    reduced_divisor m;
    int q;
    __host__ 
    fused_preop(int _a, int _c, int _m, int _q) : a(_a) , c(_c), m(_m), q(_q) {}
    __host__ __device__ __forceinline__
    int p(const int& i) {
        int cm1 = (int)c.get() - 1;
        int term_1 = int(a.get()) * (int)c.mod(cm1 * i);
        int term_2 = int(a.mod(int(c.div(cm1+i))*q));
        return term_1 + term_2;
        
    }
    __host__ __device__
    int operator()(const int& i, const int& j) {
        int idx = m.mod(i + (int)m.get() - (int)m.mod(j));
        return p(idx);
    }
};

struct fused_postop {
    reduced_divisor m;
    reduced_divisor b;
    __host__  fused_postop(int _m, int _b) : m(_m), b(_b) {}
    __host__ __device__
    int operator()(const int& i, const int& j) {
        return (int)m.mod(i + (int)m.get() - (int)b.div(j));
    }
};


}


template<typename T, typename F, int U>
__global__ void long_row_shuffle(int m, int n, int i, T* d, T* tmp, F s) {
    row_major_index rm(m, n);
    s.set_i(i);
    int global_id = threadIdx.x + blockIdx.x * blockDim.x;
    int grid_size = gridDim.x * blockDim.x;
    int j = global_id;
    while(j + U * grid_size < n) {
        #pragma unroll
        for(int k = 0; k < U; k++) {
            tmp[j] = d[rm(i, s(j))];
            j += grid_size;
        }
    }
    while(j < n) {
        tmp[j] = d[rm(i, s(j))];
        j += grid_size;
    }
}

template<typename T, typename F>
__global__ void short_column_permute(int m, int n, T* d, F s) {
    T* smem = shared_memory<T>();
    row_major_index rm(m, n);
    row_major_index blk(blockDim.y, blockDim.x);
    int i = threadIdx.y; // One block tall by REQUIREMENT
    int grid_size = blockDim.x * gridDim.x;
    
    if (i < m) {
        for(int j = threadIdx.x + blockIdx.x * blockDim.x;
            j < n; j+= grid_size) {
            
            smem[blk(i, threadIdx.x)] = d[rm(i, j)];
            __syncthreads();
            d[rm(i, j)] = smem[blk(s(i, j), threadIdx.x)];
            __syncthreads();

        }   
    }
}

template<typename T, typename F>
void skinny_row_op(F s, int m, int n, T* d, T* tmp) {
    for(int i = 0; i < m; i++) {
        long_row_shuffle<T, F, 4><<<(n-1)/(256*4)+1,256>>>(m, n, i, d, tmp, s);
        hipMemcpy(d + n * i, tmp, sizeof(T) * n, hipMemcpyDeviceToDevice);

    }
}

template<typename T, typename F>
void skinny_col_op(F s, int m, int n, T* d) {
    int n_threads = 32;
    // XXX Potential optimization here: figure out how many blocks/sm
    // we should launch
    int n_blocks = n_sms()*8;
    dim3 grid_dim(n_blocks);
    dim3 block_dim(n_threads, m);
    short_column_permute<<<grid_dim, block_dim,
        sizeof(T) * m * n_threads>>>(m, n, d, s);
}


namespace c2r {

template<typename T>
void skinny_transpose(T* data, int m, int n) {
    //std::cout << "Doing Skinny C2R transpose of " << m << ", " << n << std::endl;

    assert(m <= 32);
    int c, t, k;
    extended_gcd(m, n, c, t);
    if (c > 1) {
        extended_gcd(m/c, n/c, t, k);
    } else {
        k = t;
    }

    if (c > 1) {
        skinny_col_op(fused_preop(m, n/c), m, n, data);
    }
    T* tmp;
    hipMalloc(&tmp, sizeof(T) * n);
    skinny_row_op(long_shuffle(m, n, c, k), m, n, data, tmp);
    hipFree(tmp);
    skinny_col_op(fused_postop(m, n, c), m, n, data);

}


template void skinny_transpose(float* data, int m, int n);
template void skinny_transpose(double* data, int m, int n);
template void skinny_transpose(int* data, int m, int n);
template void skinny_transpose(long long* data, int m, int n);

}

namespace r2c {

template<typename T>
void skinny_transpose(T* data, int m, int n) {
    //std::cout << "Doing Skinny R2C transpose of " << m << ", " << n << std::endl;

    assert(m <= 32);
    int c, t, q;
    extended_gcd(n, m, c, t);
    if (c > 1) {
        extended_gcd(n/c, m/c, t, q);
    } else {
        q = t;
    }

    skinny_col_op(fused_preop(m/c, c, m, q), m, n, data);
    T* tmp;
    hipMalloc(&tmp, sizeof(T) * n);
    skinny_row_op(shuffle(m, n, c, 0), m, n, data, tmp);
    hipFree(tmp);
    if (c > 1) {
        skinny_col_op(fused_postop(m, n/c), m, n, data);
    }
}

template void skinny_transpose(float* data, int m, int n);
template void skinny_transpose(double* data, int m, int n);
template void skinny_transpose(int* data, int m, int n);
template void skinny_transpose(long long* data, int m, int n);

}



}
}
